
#include <hip/hip_runtime.h>


class CudaConvertible
{
public:
	template<typename T>
	static T* ConvertPtr(T* value, T* destination){
		  hipError_t cudaStatus;
	
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		cudaStatus = 
			hipMalloc((void**)&destination, sizeof(T));
		if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
       // goto Error;
    }

	  cudaStatus = hipMemcpy(destination, value, sizeof(T), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
	}	
	return destination;
	};

	
};