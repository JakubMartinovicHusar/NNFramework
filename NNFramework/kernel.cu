#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <cstdlib>
#include <ctime>
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "Data.cuh"
#include "Network.cuh"



hipError_t trainCycleWithCuda(Network *network);//, DataList *data, int population_size);

__global__ void trainCycleKernel(Network *network)//, DataList *data, int *population_size)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	/*for(int m = 0; m<data->_datalist_size; m++){
		Network n = network[i];//.Calculate(&data[m]);
		float* results = n.Calculate(&(data->data[m]), data->_input_size,1);
		delete[] results;
	}*/
}

int main()
{

	int population_size = 100;
	int data_size = 1000;
	Network* network = new Network[population_size];
	const int input_size = 3;
	DataList* data = new DataList(data_size, input_size);
	for(int i = 0; i<data_size; i++) {
		float inputs[input_size];
		for(int j = 0; j<input_size; j++) {
			inputs[j] = static_cast<float>(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
		}
		data->AddData(inputs, i);
	}
	// Add vectors in parallel.
	hipError_t cudaStatus = trainCycleWithCuda(network);//, data, population_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	system("pause");

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t trainCycleWithCuda(Network *network)//, DataList *data, int population_size)
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	Network *network_dev;
	//DataList *data_dev;
	//Data* datavalues_dev;
	/*  int *dev_a = 0;*/
	int *population_size_dev = 0;
	network[0].Initialize();

	network_dev = network[0].GetCudaCopy();
	/*int *data_size_dev = 0;
	


	// Choose which GPU to run on, change this on a multi-GPU system.


	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}



	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&population_size_dev, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(population_size)!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&data_size_dev, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(data_size)!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **) &network_dev, population_size * sizeof(Network));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(network)");
		goto Error;
	}

	cudaStatus = hipMalloc((void **) &data_dev, sizeof(DataList)/*+sizeof(Data)*data->_datalist_size*/
	/*);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(data)!");
		goto Error;
	}


	cudaStatus = hipMemcpy(network_dev, network, population_size *  sizeof(Network), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed(network)!");
		goto Error;
	}


	cudaStatus = hipMemcpy(data_dev, data, sizeof(DataList)/*+sizeof(Data)*data->_datalist_size*//*, hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed(network)!");
		goto Error;
	}
	*/
	/*
	//inputs = new float[data->_input_size];
	Data * dl = new Data[data->_datalist_size];
	for (size_t i = 0; i < data->_datalist_size; ++i)
	{
		float * ild;
		/*float * il = (float *)malloc(data->_input_size*sizeof(float));
		for(size_t j = 0; j < data->_input_size; ++j){
			il[j] = data->data[i].inputs[j];
		}
		hipMalloc((void**) &ild, data->_input_size*sizeof(float));
		hipMemcpy(ild, data->data[i].inputs, data->_input_size*sizeof(float), hipMemcpyHostToDevice);

		Data * d;
		cudaStatus = hipMalloc((void **) &d, sizeof(Data));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed(data)!");
			goto Error;
		}
		cudaStatus = hipMemcpy(d, &(data->data)[i], sizeof(Data), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed(network)!");
			goto Error;
		}
		d->inputs = ild;
		dl[i] = *d; 
		delete[] ild;
	}
	data_dev->data = dl;



//data_dev->data = &datavalues_dev;


cudaStatus = hipMemcpy(population_size_dev, &population_size,  sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed(population_size)!");
	goto Error;
}
	*/

// Launch a kernel on the GPU with one thread for each element.
int numBlocks = 1;
dim3 threadsPerBlock((unsigned int)1,1);
trainCycleKernel<<<numBlocks, threadsPerBlock>>>(network_dev);//, data_dev, population_size_dev);
/*
// Check for any errors launching the kernel
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
	fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	goto Error;
}

// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	goto Error;
}
*/
// Copy output vector from GPU buffer to host memory.
/* cudaStatus = hipMemcpy(network_dev, network, sizeof(network), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}*/
/*
Error:
hipFree(network_dev);
*/

return cudaStatus;
}
